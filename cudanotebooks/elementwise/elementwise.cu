#include "hip/hip_runtime.h"
// https://github.com/xlite-dev/CUDA-Learn-Notes/blob/main/kernels/elementwise/elementwise.cu
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
// #include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>

#include <torch/types.h>
// #include <torch/extension.h>

#include <cuda_learning_utils.h>

#define WARP_SIZE 32

/**
 * reinterpret_cast: 强制转换编译时对变量位模式的解释类型（对其底层二进制代码的解释）。如：
 * ```cpp
 * int num=6513249; // 6513249(10) = 0x00636261(16)
 * int *pnum=&num; 
 * // pnum是int指针，int为4字节。位模式下 *pnum 会直接读取指针指向地址的连续4字节的数据，再翻译成int，即6513249。
 * 
 * char *pstr = reinterpret_cast<char*>pnum;
 * // pstr是 char指针，char为1字节。位模式下 *pstr 会直接读取指针指向地址的连续1字节的数据。
 * 因为是小端存储，所以从右往左读十六进制的两位数8个比特，即0x61，再根据ASCII码翻译得 a。
 * ```
 * 详见`cppnotebooks/reinterpret_cast.cc`
 */
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])


// -------------------------------------- FP32 -------------------------------------- 
// ElementWise Add  
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_fp32_kernel(const float *a, const float *b, float *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<N) c[idx]=a[idx]+b[idx];
}

/**
 * 2-dim torch::Tensor saves elements in mem is Row-Major Order(行优先),
 * so it can be elementwise-added in cuda in one-dim.
 */
torch::Tensor elementwise_add_fp32(const torch::Tensor &a,const torch::Tensor &b)
{
    CHECK_TORCH_TENSOR_DEVICE(a);
    CHECK_TORCH_TENSOR_DEVICE(b);
    CHECK_TORCH_TENSORS_SIZES(a,b);
    
    const int ndim = a.dim();
    const int64_t N = a.numel();
    torch::Tensor output = torch::empty_like(a);

    dim3 block_size(16);
    dim3 grid_size((N+block_size.x-1)/block_size.x);
    elementwise_add_fp32_kernel<<<grid_size, block_size>>>(
        a.data_ptr<float>(),
        b.data_ptr<float>(),
        output.data_ptr<float>(),
        N
    );
    CUDA_ERROR_LOG

    return output;
}

// --------------------- PyTorch bindings for custom kernel -----------------------

TORCH_PYBIND(elementwise_add_fp32)

// TORCH_LIBRARY(custom_ops, m){m.def("add_fp32",torch_add_fp32);}